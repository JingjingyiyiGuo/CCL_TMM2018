#include <vector>

#include "gtest/gtest.h"

#include "caffe/common_layers.hpp"
#include "caffe/filler.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void FuseInnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  //printf("FuseInnerProduce_Forward_gpu\n");
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data_1 = top[0]->mutable_gpu_data();
  Dtype* top_data_2 = top[1]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weight, bottom_data, (Dtype)0., top_data_1);
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weight, bottom_data, (Dtype)0., top_data_2);
    // top_data_2 = top_data_1
    //caffe_gpu_axpby<Dtype>(N_, (Dtype)1., top_data_2, (Dtype)0., top_data_1);
    if (bias_term_) {
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            this->blobs_[1]->gpu_data(), top_data_1);
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            this->blobs_[1]->gpu_data(), top_data_2);
      //caffe_copy<Dtype>(N_,top_data_1,top_data_2);
      //caffe_gpu_axpby<Dtype>(N_, (Dtype)1., top_data_2, (Dtype)0., top_data_1);
    }
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
                          bottom_data, weight, (Dtype)0., top_data_1);
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
                          bottom_data, weight, (Dtype)0., top_data_2);
    //caffe_copy<Dtype>(N_,top_data_1,top_data_2);
    //caffe_gpu_axpby<Dtype>(N_, (Dtype)1., top_data_2, (Dtype)0., top_data_1);
    if (bias_term_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            this->blobs_[1]->gpu_data(), (Dtype)1., top_data_1);
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            this->blobs_[1]->gpu_data(), (Dtype)1., top_data_2);
      //caffe_copy<Dtype>(N_,top_data_1,top_data_2);
      //caffe_gpu_axpby<Dtype>(N_, (Dtype)1., top_data_2, (Dtype)0., top_data_1);
    }
  }
}

template <typename Dtype>
void FuseInnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  //printf("FuseInnerProduce_Backward_gpu\n");
  if (this->param_propagate_down_[0]) {
    // sum two top_diff
    const Dtype* top_diff_1 = top[0]->gpu_diff();
    const Dtype* top_diff_2 = top[1]->gpu_diff();
    //caffe_gpu_axpy<Dtype>(N_, (Dtype)1., top_diff_1, top_diff);

    const Dtype* bottom_data = bottom[0]->gpu_data();
    // Gradient with respect to weight
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
        top_diff_1, bottom_data, (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
        top_diff_2, bottom_data, (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff_1 = top[0]->gpu_diff();
    const Dtype* top_diff_2 = top[1]->gpu_diff();
    //caffe_gpu_axpy<Dtype>(N_, (Dtype)1., top_diff_1, top_diff);
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff_1,
        bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff_2,
        bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
  }
  if (propagate_down[0]) {
    const Dtype* top_diff_1 = top[0]->gpu_diff();
    const Dtype* top_diff_2 = top[1]->gpu_diff();
    //caffe_gpu_axpy<Dtype>(N_, (Dtype)1., top_diff_1, top_diff);
    // Gradient with respect to bottom data
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
        top_diff_1, this->blobs_[0]->gpu_data(), (Dtype)0.,
        bottom[0]->mutable_gpu_diff());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
        top_diff_2, this->blobs_[0]->gpu_data(), (Dtype)1.,
        bottom[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(FuseInnerProductLayer);

}  // namespace caffe
